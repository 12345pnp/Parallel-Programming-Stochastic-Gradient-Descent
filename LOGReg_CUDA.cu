#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>


#define SZ 768
#define TOTITER 100000000
#define THRDS 1
//#define MAXITER TOTITER/THRDS

void checkCUDAError(const char* msg);

__host__ __device__ float Y_Model(float W1,float W2,float B,float X1,float X2){

	float Z = B + W1*X1 + W2*X2; 
	return 1.0/( 1.0 + exp(-Z));

}

__global__ void gd(float *X1,float *X2,float *Y,float *W1,float *W2,float *B){

	float h = 0.0001,y,dW1,dW2,dB;

    int idx;
    int MAXITER =  TOTITER/THRDS;
    unsigned int XX = 562628;
    unsigned int a = 1212*(threadIdx.x+1);
    unsigned int c = 3238 + (threadIdx.x+1);
    unsigned int m = 8191211;


    for(int i = 0; i<MAXITER; ++i)
    {
    	XX = (a*XX + c)%(m*(i+1)*(threadIdx.x+1)); //Linear Conguential Pseudo-Random Number Generator
        
        idx = XX%SZ;

        y = Y_Model(*W1,*W2,*B, X1[idx], X2[idx]);

	    dW1 = h*(Y[idx] - y)*y*(1.0 - y)*X1[idx];	    
	    dW2 = h*(Y[idx] - y)*y*(1.0 - y)*X2[idx];
		dB  =  h*(Y[idx] - y)*y*(1.0 - y);
         
        atomicAdd(W1, dW1);

        atomicAdd(W2, dW2);
         
        atomicAdd(B, dB);

    }

}

int main(){

    struct timeval start, end;
    srand (time(NULL));

	int numthreads=THRDS;
	int numblocks=1;
	
	float X[768][9];
	FILE *fp;


	float *h_X1;
	h_X1 = (float*)malloc(SZ*sizeof(float));
	float *h_X2;
	h_X2 = (float*)malloc(SZ*sizeof(float));

	float *h_Y;
	h_Y = (float*)malloc(SZ*sizeof(float));

	float *h_W1;
	h_W1 = (float*)malloc(sizeof(float));

	float *h_W2;
	h_W2 = (float*)malloc(sizeof(float));
	float *h_B;
	h_B = (float*)malloc(sizeof(float));

    *h_W1 = 0;
	*h_W2 = 0;
    *h_B = 0;
    
    float *d_X1, *d_X2, *d_Y, *d_W1, *d_W2, *d_B;

	hipMalloc((void**)&d_X1,SZ*sizeof(float));

	hipMalloc((void**)&d_X2,SZ*sizeof(float));

	hipMalloc((void**)&d_Y,SZ*sizeof(float));

	hipMalloc((void**)&d_W1,sizeof(float));

    hipMalloc((void**)&d_W2,sizeof(float));

    hipMalloc((void**)&d_B,sizeof(float));

	fp=fopen("input.txt","r");

	for(int i=0;i<SZ;i++){
		char *buff=(char*) malloc(70);

		fgets(buff, 70, fp);

		int count=0;

		int j=0;

		while(count<9){
			char *c=(char*) malloc(50);
			int l = 0;   
			while(buff[j]!=',' && buff[j]!='\0')
			{
				c[l] = buff[j];
				j++; l++; 

			}
			X[i][count] = atof(c);
			free (c);
			count++;
			if(count<9)
				j++;
		}

	}

	for(int i=0;i<SZ;i++)
	{
      h_X1[i] = X[i][2];
      h_X2[i] = X[i][5];
      h_Y[i] = X[i][8];
	}

	fclose(fp);

	hipMemcpy(d_X1,h_X1,SZ*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_X2,h_X2,SZ*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_Y,h_Y,SZ*sizeof(float),hipMemcpyHostToDevice);


	hipMemcpy(d_W1,h_W1,sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_W2,h_W2,sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,sizeof(float),hipMemcpyHostToDevice);
	
    gettimeofday(&start,NULL);

	gd<<<numblocks,numthreads>>>(d_X1,d_X2, d_Y,d_W1,d_W2,d_B);
    hipDeviceSynchronize();

	checkCUDAError("kernel invocation");
	gettimeofday(&end,NULL);


    hipMemcpy(h_W1, d_W1, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_W2, d_W2, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, sizeof(float), hipMemcpyDeviceToHost);

     float error = 0;

for(int i =0; i<SZ; ++i)
{
	error += pow((h_Y[i] - Y_Model(*h_W1,*h_W2,*h_B, h_X1[i], h_X2[i]) ),2);
}

error = sqrt(error);

error = error/SZ;

//int k = 10;
    std::cout<<"error "<<error<<'\n';
    
    printf("W1 = %f W2 = %f B = %f\n", *h_W1, *h_W2, *h_B);

    std::cout<<"Number of Threads: "<<numthreads<<'\n';
    std::cout<<"Total Number of Steps: "<<TOTITER<<'\n';
    std::cout<<"Time taken: \n"<<(end.tv_sec - start.tv_sec)*1000000 + end.tv_usec - start.tv_usec<<" microseconds. \n"; 

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
		exit(EXIT_FAILURE);
	}
}

